
#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void addThreadId() 
{
	int i = threadIdx.x;


	printf("Hello world! My threadId is %d\n",i);
}

int main()
{
	const int threads = 256;
	addThreadId <<<1, threads >>> ();


	hipDeviceSynchronize();
    return 0;
}